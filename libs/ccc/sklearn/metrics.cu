#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cmath>
#include <assert.h>


// #define N_OBJS 16
// #define N_PARTS 1
// #define N_FEATURES 2


/**
 * @brief Unravel a flat index to the corresponding 2D indicis
 * @param[in] flat_idx The flat index to unravel
 * @param[in] num_cols Number of columns in the 2D array
 * @param[out] row Pointer to the row index
 * @param[out] col Pointer to the column index
 */
__device__ __host__ inline void unravel_index(int flat_idx, int num_cols, int* row, int* col) {
    // change int to uint32_t
    *row = flat_idx / num_cols;  // Compute row index
    *col = flat_idx % num_cols;  // Compute column index
}


__device__ __host__ inline void get_coords_from_index(int n_obj, int idx, int* x, int* y) {
    // Calculate 'b' based on the input n_obj
    int b = 1 - 2 * n_obj;
    // Calculate 'x' using the quadratic formula part
    float discriminant = b * b - 8 * idx;
    float x_float = floor((-b - sqrt(discriminant)) / 2);
    // Assign the integer part of 'x'
    *x = static_cast<int>(x_float);
    // Calculate 'y' based on 'x' and the index
    *y = static_cast<int>(idx + (*x) * (b + (*x) + 2) / 2 + 1);
}


/**
 * @brief Main ARI kernel. Now only compare a pair of ARIs
 * @param n_parts Number of partitions of each feature
 * @param n_objs Number of objects in each partitions
 * @param n_part_mat_elems Number of elements in the square partition matrix
 * @param n_elems_per_feat Number of elements for each feature, i.e., part[i].x * part[i].y
 * @param parts 3D Array of partitions with shape of (n_features, n_parts, n_objs)
 * @param uniqs Array of unique counts
 * @param n_aris Number of ARIs to compute
 * @param out Output array of ARIs
 * @param part_pairs Output array of part pairs to be compared by ARI
 */
__global__
void ari(int* parts,
         int* uniqs,
         const int n_aris,
         const int n_features,
         const int n_parts,
         const int n_objs,
         const int n_elems_per_feat,
         const int n_part_mat_elems,
         float* out,
         int* part_pairs = nullptr
         )
{
    /*
    * Step 1: Each thead, unravel flat indices and load the corresponding data into shared memory
    */
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    // each block is responsible for one ARI computation
    int ari_block_idx = blockIdx.x;

    // print parts for debugging
    if (global_tid == 0) {
        for (int i = 0; i < n_features; ++i) {
            for (int j = 0; j < n_parts; ++j) {
                for (int k = 0; k < n_objs; ++k) {
                    printf("parts[%d][%d][%d]: %d\n", i, j, k, parts[i * n_parts * n_objs + j * n_objs + k]);
                }
            }
            printf("\n");
        }
    }

    // obtain the corresponding parts and unique counts
    printf("n_part_mat_elems: %d\n", n_part_mat_elems);
    int feature_comp_flat_idx = ari_block_idx / n_part_mat_elems;   // flat comparison pair index for two features
    int part_pair_flat_idx = ari_block_idx % n_part_mat_elems;  // flat comparison pair index for two partitions of one feature pair
    int i, j;

    if (global_tid == 0) {
        printf("ari_block_idx: %d, feature_comp_flat_idx: %d, part_pair_flat_idx: %d\n", ari_block_idx, feature_comp_flat_idx, part_pair_flat_idx);
    }

    // unravel the feature indices
    get_coords_from_index(n_features, feature_comp_flat_idx, &i, &j);
    assert(i < n_features && j < n_features);
    assert(i >= 0 && j >= 0);
    if (global_tid == 0) {
        printf("global_tid: %d, i: %d, j: %d\n", global_tid, i, j);
    }
    // unravel the partition indices
    int m, n;
    unravel_index(part_pair_flat_idx, n_parts, &m, &n);
    if (global_tid == 0){
        printf("global_tid: %d, m: %d, n: %d\n", global_tid, m, n);
    }
    
    // Make pointers to select the parts and unique counts for the feature pair
    // Todo: Use int4*?
    int* t_data_part0 = parts + i * n_elems_per_feat + m * n_objs ;  // t_ for thread
    int* t_data_part1 = parts + j * n_elems_per_feat + n * n_objs ;
    //int* t_data_uniqi = uniqs + i * n_parts + m;
    //int* t_data_uniqj = uniqs + j * n_parts + n;
    
    // Load gmem data into smem by using different threads
    extern __shared__ int shared_mem[];
    int* s_part0 = shared_mem;
    int* s_part1 = shared_mem + n_objs;
    
    // Loop over the data using the block-stride pattern
    for (int i = threadIdx.x; i < n_objs; i += blockDim.x) {
        s_part0[i] = t_data_part0[i];
        s_part1[i] = t_data_part1[i];
    }
    __syncthreads();

    // Copy data to global memory if part_pairs is specified
    if (part_pairs != nullptr) {
        int* out_part0 = part_pairs + ari_block_idx * (2 * n_objs);
        int* out_part1 = out_part0 + n_objs;

        for (int i = threadIdx.x; i < n_objs; i += blockDim.x) {
            out_part0[i] = s_part0[i];
            out_part1[i] = s_part1[i];
        }
    }
    
    /*
    * Step 2: Compute contingency matrix within the block
    */


    /*
    * Step 3: Construct pair confusion matrix
    */

    /*
    * Step 4: Compute ARI and write to global memory
    */
}

// Helper function to generate pairwise combinations (implement this according to your needs)
std::vector<std::pair<std::vector<int>, std::vector<int>>> generate_pairwise_combinations(const std::vector<std::vector<std::vector<int>>>& arr) {
    std::vector<std::pair<std::vector<int>, std::vector<int>>> pairs;
    size_t num_slices = arr.size();  // Number of 2D arrays in the 3D vector
    for (size_t i = 0; i < num_slices; ++i) {
        for (size_t j = i + 1; j < num_slices; ++j) {  // Only consider pairs in different slices
            for (const auto& row_i : arr[i]) {  // Each row in slice i
                for (const auto& row_j : arr[j]) {  // Pairs with each row in slice j
                    pairs.emplace_back(row_i, row_j);
                }
            }
        }
    }
    return pairs;
}

void test_ari_parts_selection() {
    // Define test input
    std::vector<std::vector<std::vector<int>>> parts = {
        {{11, 12, 23, 34},
         {12, 23, 34, 45},
         {13, 34, 45, 56}},
        {{21, 12, 23, 34},
         {22, 23, 34, 45},
         {23, 34, 45, 56}},
        {{31, 12, 23, 34},
         {32, 23, 34, 45},
         {33, 34, 45, 56}}
    };


    // Get dimensions
    int n_features = parts.size();
    int n_parts = parts[0].size();
    int n_objs = parts[0][0].size();
    int n_feature_comp = n_features * (n_features - 1) / 2;
    int n_aris = n_feature_comp * n_parts * n_parts;
    std::cout << "n_features: " << n_features << ", n_parts: " << n_parts << ", n_objs: " << n_objs << std::endl << "n_feature_comps: " << n_feature_comp <<  ", n_aris: " << n_aris << std::endl;

    // Allocate host memory for C-style array
    int* h_parts = new int[n_features * n_parts * n_objs];

    // Copy data from vector to C-style array
    for (int i = 0; i < n_features; ++i) {
        for (int j = 0; j < n_parts; ++j) {
            for (int k = 0; k < n_objs; ++k) {
                h_parts[i * (n_parts * n_objs) + j * n_objs + k] = parts[i][j][k];
            }
        }
    }

    // Set up CUDA kernel configuration
    int block_size = 2;
    // Each block is responsible for one ARI computation
    int grid_size = n_aris;
    size_t s_mem_size = n_objs * 2 * sizeof(int);

    // Allocate device memory
    int *d_parts, *d_uniqs, *d_parts_pairs;
    float *d_out;
    hipMalloc(&d_parts, n_features * n_parts * n_objs * sizeof(int));
    hipMalloc(&d_uniqs, n_objs * sizeof(int));
    hipMalloc(&d_out, n_aris * sizeof(float));
    hipMalloc(&d_parts_pairs, n_aris * 2 * n_objs * sizeof(int));

    // Copy data to device
    hipMemcpy(d_parts, h_parts, n_features * n_parts * n_objs * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    ari<<<grid_size, block_size, s_mem_size>>>(
        d_parts,
        d_uniqs,
        n_aris,
        n_features,
        n_parts,
        n_objs,
        n_parts * n_objs,
        n_parts * n_parts,
        d_out,
        d_parts_pairs
    );

    // Synchronize device
    hipDeviceSynchronize();

    // Copy results back to host
    int* h_parts_pairs = new int[n_aris * 2 * n_objs];
    hipMemcpy(h_parts_pairs, d_parts_pairs, n_aris * 2 * n_objs * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    std::cout << "Parts pairs: " << std::endl;
    for (int i = 0; i < n_aris; ++i) {
        std::cout << "Pair:" << i << std::endl;
        for (int j = 0; j < 2; ++j) {
            for (int k = 0; k < n_objs; ++k) {
                std::cout << *(h_parts_pairs + i * 2 * n_objs + j * n_objs + k) << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl << std::endl;
    }
    std::cout << std::endl;

    // Assert equality on the parts pairs
    bool all_equal = true;
    auto pairs = generate_pairwise_combinations(parts);
    int n_pairs = pairs.size();
    for (int i = 0; i < n_pairs; ++i) {
        for (int j = 0; j < 2; ++j) {
            const std::vector<int>& current_vector = (j == 0) ? pairs[i].first : pairs[i].second;
            for (int k = 0; k < n_objs; ++k) {
                int flattened_index = i * 2 * n_objs + j * n_objs + k;
                if (h_parts_pairs[flattened_index] != current_vector[k]) {
                    all_equal = false;
                    std::cout << "Mismatch at i=" << i << ", j=" << j << ", k=" << k << std::endl;
                    std::cout << "Expected: " << current_vector[k] << ", Got: " << h_parts_pairs[flattened_index] << std::endl;
                }
            }
        }
    }

    if (all_equal) {
        std::cout << "Test passed: All elements match." << std::endl;
    } else {
        std::cout << "Test failed: Mismatches found." << std::endl;
    }

    // Clean up
    hipFree(d_parts);
    hipFree(d_uniqs);
    hipFree(d_out);
    hipFree(d_parts_pairs);
    delete[] h_parts_pairs;
}

int main() {
    test_ari_parts_selection();
    return 0;
}